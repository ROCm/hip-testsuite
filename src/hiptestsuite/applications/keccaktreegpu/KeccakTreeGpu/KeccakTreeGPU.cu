#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
GPU Implementation of Keccak by Guillaume Sevestre, 2010

This code is hereby put in the public domain.
It is given as is, without any guarantee.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>


//#include "KeccakTreeGPU.h"

#include "KeccakTree.h"
#include "KeccakF.h"


extern "C" 
__host__ void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

//GPU constants
 __constant__ tKeccakLane KeccakF_RoundConstants[22] =
{
    (tKeccakLane)0x00000001 ,
    (tKeccakLane)0x00008082 ,
    (tKeccakLane)0x0000808a ,
    (tKeccakLane)0x80008000 ,
    (tKeccakLane)0x0000808b ,
    (tKeccakLane)0x80000001 ,
    (tKeccakLane)0x80008081 ,
    (tKeccakLane)0x00008009 ,
    (tKeccakLane)0x0000008a ,
    (tKeccakLane)0x00000088 ,
    (tKeccakLane)0x80008009 ,
    (tKeccakLane)0x8000000a ,
    (tKeccakLane)0x8000808b ,
    (tKeccakLane)0x0000008b ,
    (tKeccakLane)0x00008089 ,
    (tKeccakLane)0x00008003 ,
    (tKeccakLane)0x00008002 ,
    (tKeccakLane)0x00000080 ,
    (tKeccakLane)0x0000800a ,
    (tKeccakLane)0x8000000a ,
    (tKeccakLane)0x80008081 ,
    (tKeccakLane)0x00008080
};

//host constants
 tKeccakLane KeccakF_RoundConstants_h[22] =
{
    (tKeccakLane)0x00000001 ,
    (tKeccakLane)0x00008082 ,
    (tKeccakLane)0x0000808a ,
    (tKeccakLane)0x80008000 ,
    (tKeccakLane)0x0000808b ,
    (tKeccakLane)0x80000001 ,
    (tKeccakLane)0x80008081 ,
    (tKeccakLane)0x00008009 ,
    (tKeccakLane)0x0000008a ,
    (tKeccakLane)0x00000088 ,
    (tKeccakLane)0x80008009 ,
    (tKeccakLane)0x8000000a ,
    (tKeccakLane)0x8000808b ,
    (tKeccakLane)0x0000008b ,
    (tKeccakLane)0x00008089 ,
    (tKeccakLane)0x00008003 ,
    (tKeccakLane)0x00008002 ,
    (tKeccakLane)0x00000080 ,
    (tKeccakLane)0x0000800a ,
    (tKeccakLane)0x8000000a ,
    (tKeccakLane)0x80008081 ,
    (tKeccakLane)0x00008080
};

// Device (GPU) Keccak-f function implementation
// unrolled
__device__ void KeccakFunr( tKeccakLane * state )
{
    unsigned int round; //try to avoid to many registers
    tKeccakLane BC[5];
    tKeccakLane temp;

    for ( round = 0; round < cKeccakNumberOfRounds; ++round )
    {

        {
            // Theta
            BC[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
            BC[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
            BC[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
            BC[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
            BC[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];

            temp = BC[4] ^ ROL32(BC[1], 1);//x=0
            state[0] ^= temp;
            state[5] ^= temp;
            state[10] ^= temp;
            state[15] ^= temp;
            state[20] ^= temp;
            temp = BC[0] ^ ROL32(BC[2], 1);//x=1
            state[1] ^= temp;
            state[6] ^= temp;
            state[11] ^= temp;
            state[16] ^= temp;
            state[21] ^= temp;
            temp = BC[1] ^ ROL32(BC[3], 1);//x=2
            state[2] ^= temp;
            state[7] ^= temp;
            state[12] ^= temp;
            state[17] ^= temp;
            state[22] ^= temp;
            temp = BC[2] ^ ROL32(BC[4], 1);//x=3
            state[3] ^= temp;
            state[8] ^= temp;
            state[13] ^= temp;
            state[18] ^= temp;
            state[23] ^= temp;
            temp = BC[3] ^ ROL32(BC[0], 1);//x=4
            state[4] ^= temp;
            state[9] ^= temp;
            state[14] ^= temp;
            state[19] ^= temp;
            state[24] ^= temp;
        }//end Theta

        {
            // Rho Pi
            temp = state[1];
            BC[0] = state[10];
            state[10] = ROL32( temp, 1);
            temp = BC[0];//x=0
            BC[0] =  state[7];
            state[7] = ROL32( temp, 3);
            temp = BC[0];
            BC[0] = state[11];
            state[11] = ROL32( temp, 6);
            temp = BC[0];
            BC[0] = state[17];
            state[17] = ROL32( temp,10);
            temp = BC[0];
            BC[0] = state[18];
            state[18] = ROL32( temp,15);
            temp = BC[0];
            BC[0] =  state[3];
            state[3] = ROL32( temp,21);
            temp = BC[0];//x=5
            BC[0] =  state[5];
            state[5] = ROL32( temp,28);
            temp = BC[0];
            BC[0] = state[16];
            state[16] = ROL32( temp, 4);
            temp = BC[0];
            BC[0] =  state[8];
            state[8] = ROL32( temp,13);
            temp = BC[0];
            BC[0] = state[21];
            state[21] = ROL32( temp,23);
            temp = BC[0];
            BC[0] = state[24];
            state[24] = ROL32( temp, 2);
            temp = BC[0];//x=10
            BC[0] =  state[4];
            state[4] = ROL32( temp,14);
            temp = BC[0];
            BC[0] = state[15];
            state[15] = ROL32( temp,27);
            temp = BC[0];
            BC[0] = state[23];
            state[23] = ROL32( temp, 9);
            temp = BC[0];
            BC[0] = state[19];
            state[19] = ROL32( temp,24);
            temp = BC[0];
            BC[0] = state[13];
            state[13] = ROL32( temp, 8);
            temp = BC[0];//x=15
            BC[0] = state[12];
            state[12] = ROL32( temp,25);
            temp = BC[0];
            BC[0] =  state[2];
            state[2] = ROL32( temp,11);
            temp = BC[0];
            BC[0] = state[20];
            state[20] = ROL32( temp,30);
            temp = BC[0];
            BC[0] = state[14];
            state[14] = ROL32( temp,18);
            temp = BC[0];
            BC[0] = state[22];
            state[22] = ROL32( temp, 7);
            temp = BC[0];//x=20
            BC[0] =  state[9];
            state[9] = ROL32( temp,29);
            temp = BC[0];
            BC[0] =  state[6];
            state[6] = ROL32( temp,20);
            temp = BC[0];
            BC[0] =  state[1];
            state[1] = ROL32( temp,12);
            temp = BC[0];//x=23
        }//end Rho Pi

        {
            //	Chi
            BC[0] = state[0];
            BC[1] = state[1];
            BC[2] = state[2];
            BC[3] = state[3];
            BC[4] = state[4];
            state[0] = BC[0] ^((~BC[1]) & BC[2]);
            state[1] = BC[1] ^((~BC[2]) & BC[3]);
            state[2] = BC[2] ^((~BC[3]) & BC[4]);
            state[3] = BC[3] ^((~BC[4]) & BC[0]);
            state[4] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[5];
            BC[1] = state[6];
            BC[2] = state[7];
            BC[3] = state[8];
            BC[4] = state[9];
            state[5] = BC[0] ^((~BC[1]) & BC[2]);
            state[6] = BC[1] ^((~BC[2]) & BC[3]);
            state[7] = BC[2] ^((~BC[3]) & BC[4]);
            state[8] = BC[3] ^((~BC[4]) & BC[0]);
            state[9] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[10];
            BC[1] = state[11];
            BC[2] = state[12];
            BC[3] = state[13];
            BC[4] = state[14];
            state[10] = BC[0] ^((~BC[1]) & BC[2]);
            state[11] = BC[1] ^((~BC[2]) & BC[3]);
            state[12] = BC[2] ^((~BC[3]) & BC[4]);
            state[13] = BC[3] ^((~BC[4]) & BC[0]);
            state[14] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[15];
            BC[1] = state[16];
            BC[2] = state[17];
            BC[3] = state[18];
            BC[4] = state[19];
            state[15] = BC[0] ^((~BC[1]) & BC[2]);
            state[16] = BC[1] ^((~BC[2]) & BC[3]);
            state[17] = BC[2] ^((~BC[3]) & BC[4]);
            state[18] = BC[3] ^((~BC[4]) & BC[0]);
            state[19] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[20];
            BC[1] = state[21];
            BC[2] = state[22];
            BC[3] = state[23];
            BC[4] = state[24];
            state[20] = BC[0] ^((~BC[1]) & BC[2]);
            state[21] = BC[1] ^((~BC[2]) & BC[3]);
            state[22] = BC[2] ^((~BC[3]) & BC[4]);
            state[23] = BC[3] ^((~BC[4]) & BC[0]);
            state[24] = BC[4] ^((~BC[0]) & BC[1]);
        }//end Chi

        //	Iota
        state[0] ^= KeccakF_RoundConstants[round];
    }

}
//end unrolled

//Host Keccak-f function (pb with using the same constants between host and device) 
//unrolled
__host__ void KeccakFunr_h( tKeccakLane * state )
{
    unsigned int round; //try to avoid to many registers
    tKeccakLane BC[5];
    tKeccakLane temp;

    for ( round = 0; round < cKeccakNumberOfRounds; ++round )
    {

        {
            // Theta
            BC[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
            BC[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
            BC[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
            BC[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
            BC[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];

            temp = BC[4] ^ ROL32(BC[1], 1);//x=0
            state[0] ^= temp;
            state[5] ^= temp;
            state[10] ^= temp;
            state[15] ^= temp;
            state[20] ^= temp;
            temp = BC[0] ^ ROL32(BC[2], 1);//x=1
            state[1] ^= temp;
            state[6] ^= temp;
            state[11] ^= temp;
            state[16] ^= temp;
            state[21] ^= temp;
            temp = BC[1] ^ ROL32(BC[3], 1);//x=2
            state[2] ^= temp;
            state[7] ^= temp;
            state[12] ^= temp;
            state[17] ^= temp;
            state[22] ^= temp;
            temp = BC[2] ^ ROL32(BC[4], 1);//x=3
            state[3] ^= temp;
            state[8] ^= temp;
            state[13] ^= temp;
            state[18] ^= temp;
            state[23] ^= temp;
            temp = BC[3] ^ ROL32(BC[0], 1);//x=4
            state[4] ^= temp;
            state[9] ^= temp;
            state[14] ^= temp;
            state[19] ^= temp;
            state[24] ^= temp;
        }//end Theta

        {
            // Rho Pi
            temp = state[1];
            BC[0] = state[10];
            state[10] = ROL32( temp, 1);
            temp = BC[0];//x=0
            BC[0] =  state[7];
            state[7] = ROL32( temp, 3);
            temp = BC[0];
            BC[0] = state[11];
            state[11] = ROL32( temp, 6);
            temp = BC[0];
            BC[0] = state[17];
            state[17] = ROL32( temp,10);
            temp = BC[0];
            BC[0] = state[18];
            state[18] = ROL32( temp,15);
            temp = BC[0];
            BC[0] =  state[3];
            state[3] = ROL32( temp,21);
            temp = BC[0];//x=5
            BC[0] =  state[5];
            state[5] = ROL32( temp,28);
            temp = BC[0];
            BC[0] = state[16];
            state[16] = ROL32( temp, 4);
            temp = BC[0];
            BC[0] =  state[8];
            state[8] = ROL32( temp,13);
            temp = BC[0];
            BC[0] = state[21];
            state[21] = ROL32( temp,23);
            temp = BC[0];
            BC[0] = state[24];
            state[24] = ROL32( temp, 2);
            temp = BC[0];//x=10
            BC[0] =  state[4];
            state[4] = ROL32( temp,14);
            temp = BC[0];
            BC[0] = state[15];
            state[15] = ROL32( temp,27);
            temp = BC[0];
            BC[0] = state[23];
            state[23] = ROL32( temp, 9);
            temp = BC[0];
            BC[0] = state[19];
            state[19] = ROL32( temp,24);
            temp = BC[0];
            BC[0] = state[13];
            state[13] = ROL32( temp, 8);
            temp = BC[0];//x=15
            BC[0] = state[12];
            state[12] = ROL32( temp,25);
            temp = BC[0];
            BC[0] =  state[2];
            state[2] = ROL32( temp,11);
            temp = BC[0];
            BC[0] = state[20];
            state[20] = ROL32( temp,30);
            temp = BC[0];
            BC[0] = state[14];
            state[14] = ROL32( temp,18);
            temp = BC[0];
            BC[0] = state[22];
            state[22] = ROL32( temp, 7);
            temp = BC[0];//x=20
            BC[0] =  state[9];
            state[9] = ROL32( temp,29);
            temp = BC[0];
            BC[0] =  state[6];
            state[6] = ROL32( temp,20);
            temp = BC[0];
            BC[0] =  state[1];
            state[1] = ROL32( temp,12);
            temp = BC[0];//x=23
        }//end Rho Pi

        {
            //	Chi
            BC[0] = state[0];
            BC[1] = state[1];
            BC[2] = state[2];
            BC[3] = state[3];
            BC[4] = state[4];
            state[0] = BC[0] ^((~BC[1]) & BC[2]);
            state[1] = BC[1] ^((~BC[2]) & BC[3]);
            state[2] = BC[2] ^((~BC[3]) & BC[4]);
            state[3] = BC[3] ^((~BC[4]) & BC[0]);
            state[4] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[5];
            BC[1] = state[6];
            BC[2] = state[7];
            BC[3] = state[8];
            BC[4] = state[9];
            state[5] = BC[0] ^((~BC[1]) & BC[2]);
            state[6] = BC[1] ^((~BC[2]) & BC[3]);
            state[7] = BC[2] ^((~BC[3]) & BC[4]);
            state[8] = BC[3] ^((~BC[4]) & BC[0]);
            state[9] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[10];
            BC[1] = state[11];
            BC[2] = state[12];
            BC[3] = state[13];
            BC[4] = state[14];
            state[10] = BC[0] ^((~BC[1]) & BC[2]);
            state[11] = BC[1] ^((~BC[2]) & BC[3]);
            state[12] = BC[2] ^((~BC[3]) & BC[4]);
            state[13] = BC[3] ^((~BC[4]) & BC[0]);
            state[14] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[15];
            BC[1] = state[16];
            BC[2] = state[17];
            BC[3] = state[18];
            BC[4] = state[19];
            state[15] = BC[0] ^((~BC[1]) & BC[2]);
            state[16] = BC[1] ^((~BC[2]) & BC[3]);
            state[17] = BC[2] ^((~BC[3]) & BC[4]);
            state[18] = BC[3] ^((~BC[4]) & BC[0]);
            state[19] = BC[4] ^((~BC[0]) & BC[1]);
            BC[0] = state[20];
            BC[1] = state[21];
            BC[2] = state[22];
            BC[3] = state[23];
            BC[4] = state[24];
            state[20] = BC[0] ^((~BC[1]) & BC[2]);
            state[21] = BC[1] ^((~BC[2]) & BC[3]);
            state[22] = BC[2] ^((~BC[3]) & BC[4]);
            state[23] = BC[3] ^((~BC[4]) & BC[0]);
            state[24] = BC[4] ^((~BC[0]) & BC[1]);
        }//end Chi

        //	Iota
        state[0] ^= KeccakF_RoundConstants_h[round];
    }

}
//end unrolled

//Keccak final node hashing results of previous nodes in sequential mode
__host__ void Keccak_top_GPU(tKeccakLane * Kstate, tKeccakLane *inBuffer , int block_number)
{
    int ind_word,k;

    for (k=0;k<block_number;k++)
    {
		
        for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
        {
            Kstate[ind_word] ^= inBuffer[ind_word + k * OUTPUT_BLOCK_SIZE_B/4];
        }
     KeccakFunr_h(Kstate);

	 //if(k%192==0)
		//printf("Kstate %08x %08x \n", Kstate[0], Kstate[1]);
    }
	//printf("\n\n");

}

//************************************************************************
//kernel implementaing hash function, hashing NB_INPUT_BLOCK (of 256 bits)
//
__global__ void ker_Keccak(tKeccakLane *d_inBuffer,tKeccakLane * d_outBuffer)
{
 
  int ind_word,k;
  tKeccakLane Kstate[25];

  //zeroize the state
  for(ind_word=0; ind_word<25; ind_word++) {Kstate[ind_word]=0; } 
  
  for (k=0;k<NB_INPUT_BLOCK;k++)
        {
            //xor input into state
            for (ind_word=0; ind_word<(INPUT_BLOCK_SIZE_B/4 ); ind_word++)
            {
                
				Kstate[ind_word] ^= 
						d_inBuffer[threadIdx.x 
						+ ind_word		* NB_THREADS 
						+ k				* NB_THREADS * INPUT_BLOCK_SIZE_B/4
						+ blockIdx.x	* NB_THREADS * INPUT_BLOCK_SIZE_B/4 * NB_INPUT_BLOCK ];
            }
            //apply GPU Keccak permutation
            KeccakFunr(Kstate);
        }

        //output hash in buffer
        for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
        {
          
	    
			d_outBuffer[threadIdx.x 
					+ ind_word *NB_THREADS
					+ blockIdx.x   *NB_THREADS * OUTPUT_BLOCK_SIZE_B/4 ]= Kstate[ind_word];
        }
}


//******************************************************
// Implementing 2 stages treehash
//******************************************************
__global__ void ker_Keccak_2stg(tKeccakLane *d_inBuffer,tKeccakLane * d_outBuffer)
{
 
  int ind_word,k;
  tKeccakLane Kstate[25];

  __shared__ tKeccakLane SharedBuffer[2 * OUTPUT_BLOCK_SIZE_B/4 * NB_THREADS];

  //zeroize the state
  for(ind_word=0; ind_word<25; ind_word++) {Kstate[ind_word]=0; } 
  
  for (k=0;k<NB_INPUT_BLOCK;k++)
        {
            //xor input into state
            for (ind_word=0; ind_word<(INPUT_BLOCK_SIZE_B/4 ); ind_word++)
            {
                
				Kstate[ind_word] ^= 
						d_inBuffer[threadIdx.x 
						+ ind_word		* NB_THREADS 
						+ k				* NB_THREADS * INPUT_BLOCK_SIZE_B/4
						+ blockIdx.x	* NB_THREADS * INPUT_BLOCK_SIZE_B/4 * NB_INPUT_BLOCK ];
            }
            //apply GPU Keccak permutation
            KeccakFunr(Kstate);
        }

        //output hash in SharedBuffer
        for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
        {
          
	    //printf("Kstate[%02u] = %08x",ind_word,Kstate[ind_word] );
			SharedBuffer[threadIdx.x 
					+ ind_word *NB_THREADS]= Kstate[ind_word];
        }
	
		//need to squeeze to produce more hash output 
		KeccakFunr(Kstate);

		for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
		{
			SharedBuffer[threadIdx.x
				+ ind_word *NB_THREADS
				+ NB_THREADS * OUTPUT_BLOCK_SIZE_B/4 ]= Kstate[ind_word];
		}

		// syncthreads should be mandatory to ensure that all writes to sharedmem is done before reading 
		__syncthreads();

		//***************
		// Second stage 

		if(threadIdx.x < NB_SCND_STAGE_THREADS )// Only first NB_SCND_STAGE_THREADS threads run this part 
		{
			//zeroize the state
			for(ind_word=0; ind_word<25; ind_word++) {Kstate[ind_word]=0; } 

			//number of input block per thread is now NB_INPUT_BLOCK_SNCD_STAGE = 2* NB_THREADS/ NB_SNCD_STAGE_THREADS
			//
			for (k=0;k<NB_INPUT_BLOCK_SNCD_STAGE;k++)
			{
				//xor input into state
				for (ind_word=0; ind_word<INPUT_BLOCK_SIZE_B/4; ind_word++)
				{	
					Kstate[ind_word] ^= 
						SharedBuffer[threadIdx.x 
						+ ind_word	* NB_SCND_STAGE_THREADS 
						+ k			* NB_SCND_STAGE_THREADS * INPUT_BLOCK_SIZE_B/4 ];
				}
				//apply Keccak permutation
				KeccakFunr(Kstate);
			}
			
			//output hash in output buffer
			for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
			{				
					
				d_outBuffer[threadIdx.x 
					+ ind_word * NB_SCND_STAGE_THREADS 
					+ blockIdx.x   * NB_SCND_STAGE_THREADS * 2*OUTPUT_BLOCK_SIZE_B/4 ]= Kstate[ind_word];
			}

			//need to squeeze to produce more hash output 
			KeccakFunr(Kstate);

			for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
			{
				
				d_outBuffer[threadIdx.x 
					+ ind_word *NB_SCND_STAGE_THREADS 
					+ NB_SCND_STAGE_THREADS * OUTPUT_BLOCK_SIZE_B/4 
					+ blockIdx.x  * NB_SCND_STAGE_THREADS * 2* OUTPUT_BLOCK_SIZE_B/4 ]= Kstate[ind_word];
			}

		}//end if threadIdx.x < NB_SCND_STAGE_THREADS 

}

//*************************
//Keccak Stream cipher mode 
// d_KeyNonce_inBuffer must point to a 256 bits Key + 256 bits Nonce (random)
//*************************
__global__ void ker_Keccak_SCipher(tKeccakLane *d_KeyNonce_inBuffer,tKeccakLane * d_outBuffer)
{

	int ind_word,k;
	tKeccakLane Kstate[25];

	//used shared memory to load only once the Key and Nonce
	__shared__ tKeccakLane Key[8]; // 8 32b words for a secret key
	__shared__ tKeccakLane Nonce[8]; //

	//load Key and Nonce in shared mem assuming that NB_THREADS used will be > 8 
	if(threadIdx.x <8)
	{
		Key[threadIdx.x]= d_KeyNonce_inBuffer[threadIdx.x];
		Nonce[threadIdx.x]=d_KeyNonce_inBuffer[threadIdx.x + 8 ];
	}
	__syncthreads();

	//zeroize the state
	for(ind_word=0; ind_word<25; ind_word++) {Kstate[ind_word]=0; } 

	//input the key
	//xor Key into state from shared mem
	for (ind_word=0; ind_word<8; ind_word++)
	{
		Kstate[ind_word] ^=Key[ind_word];
	}
	//apply GPU Keccak permutation
	KeccakFunr(Kstate);

	//input the Nonce
	//xor Key into state from shared mem
	for (ind_word=0; ind_word<8; ind_word++)
	{
		Kstate[ind_word] ^=Nonce[ind_word];
	}
	//apply GPU Keccak permutation
	KeccakFunr(Kstate);

	//input the threadIdx.x AND blockIdx.x to have different KeyStreams
	Kstate[0]^= threadIdx.x ;
	Kstate[1]^= blockIdx.x  ;
	KeccakFunr(Kstate);

	
	for (k=0;k<SC_NB_OUTPUT_BLOCK;k++)
	{
		//output KeyStreams in d_outBuffer
		for (ind_word=0; ind_word<OUTPUT_BLOCK_SIZE_B/4; ind_word++)
		{
			d_outBuffer[threadIdx.x 
				+ ind_word  * NB_THREADS
				+ k			* NB_THREADS * OUTPUT_BLOCK_SIZE_B/4
				+ blockIdx.x* NB_THREADS * OUTPUT_BLOCK_SIZE_B/4 * SC_NB_OUTPUT_BLOCK ]= Kstate[ind_word];
		}
		//Squeeze the state !
		KeccakFunr(Kstate);
	}
}




//********************************************************************************

//************************
//First Tree mode
//data to be hashed is in h_inBuffer
//output chaining values hashes are copied to h_outBuffer
//************************
extern "C" 
__host__  void KeccakTreeGPU(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
							 tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer)
{

  //copy host to device 
  hipMemcpy(d_inBuffer,h_inBuffer, INPUT_BLOCK_SIZE_B * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS, hipMemcpyHostToDevice);		
  //checkCUDAError(" Memcpy htd");

  //exec kernels 			
  
  hipLaunchKernelGGL(ker_Keccak, dim3(NB_THREADS_BLOCKS), dim3(NB_THREADS), 0, 0, d_inBuffer,d_outBuffer);
  //checkCUDAError(" ker_keccak");
  
  //hipDeviceSynchronize(); // should be useless when followed by hipMemcpy
	
  //copy back device to host	
  hipMemcpy(h_outBuffer,d_outBuffer, OUTPUT_BLOCK_SIZE_B * NB_THREADS*NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
  //checkCUDAError(" Memcpy dth");
}


//***********************************************************
//Overlap GPU kernel computation and CPU top node computation
//***********************************************************
extern "C" 
__host__  void KeccakTreeGPU_overlapCPU(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
										tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer,
										tKeccakLane *Kstate)
{

  //copy input data host to device 
  hipMemcpy(d_inBuffer,h_inBuffer, INPUT_BLOCK_SIZE_B * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS, hipMemcpyHostToDevice);		
  //checkCUDAError(" Memcpy htd");

  //execute cuda kernels (hashing data in each node) 			
  
  hipLaunchKernelGGL(ker_Keccak, dim3(NB_THREADS_BLOCKS), dim3(NB_THREADS), 0, 0, d_inBuffer,d_outBuffer);
  //checkCUDAError(" ker_keccak");
  
  //compute previous data on CPU Overlapping
  Keccak_top_GPU(Kstate,h_outBuffer,NB_THREADS*NB_THREADS_BLOCKS);
	
  //copy back device to host	
  hipMemcpy(h_outBuffer,d_outBuffer, OUTPUT_BLOCK_SIZE_B * NB_THREADS*NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
  //checkCUDAError(" Memcpy dth");
}



//***********************************************************
//Split computation over NB_STREAMS several kernels 
//***********************************************************
extern "C" 
__host__  void KeccakTreeGPU_Split(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
								   tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer)
{
	unsigned int inOffset;
	unsigned int outOffset;
	int s;

	//copy host to device 
	hipMemcpy(d_inBuffer,h_inBuffer, INPUT_BLOCK_SIZE_B * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS, hipMemcpyHostToDevice);		
	//checkCUDAError(" Memcpy htd");
	
	//Offset (for 32b Word ) 
	inOffset= INPUT_BLOCK_SIZE_B/4 * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS / NB_STREAMS ;
	outOffset = OUTPUT_BLOCK_SIZE_B/4 * NB_THREADS*NB_THREADS_BLOCKS / NB_STREAMS ;

	//exec kernels 			
	for(s=0;s<NB_STREAMS;s++)
	{
	hipLaunchKernelGGL(ker_Keccak, dim3(NB_THREADS_BLOCKS/NB_STREAMS ), dim3(NB_THREADS), 0, 0, d_inBuffer + s*inOffset ,d_outBuffer + s*outOffset);
	}//checkCUDAError(" ker_keccak");



	//copy back device to host	
	hipMemcpy(h_outBuffer,d_outBuffer, OUTPUT_BLOCK_SIZE_B * NB_THREADS*NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
	//checkCUDAError(" Memcpy dth");
}


//***********************************************************
//Split computation over several STREAMS
//***********************************************************
extern "C" 
__host__  void KeccakTreeGPU_Stream(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
								   tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer)
{
	unsigned int inOffset;
	unsigned int outOffset;
	int s;

	hipStream_t stream[NB_STREAMS];
	
	for(s=0;s<NB_STREAMS;s++)
	{hipStreamCreate(&stream[s]);}
	checkCUDAError(" StreamCreate");
		
	//Offset (for 32b Word ) 
	inOffset= INPUT_BLOCK_SIZE_B/4 * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS / NB_STREAMS ;
	outOffset = OUTPUT_BLOCK_SIZE_B/4 * NB_THREADS*NB_THREADS_BLOCKS / NB_STREAMS ;

	//exec kernels 			
	for(s=0;s<NB_STREAMS;s++)
	{
		//copy host to device, PARTIAL 
		hipMemcpyAsync(d_inBuffer + s* inOffset , h_inBuffer + s*inOffset , inOffset * 4 , hipMemcpyHostToDevice, stream[s]);		
		checkCUDAError(" MemcpyAsync htd");

		hipLaunchKernelGGL(ker_Keccak, dim3(NB_THREADS_BLOCKS/NB_STREAMS ), dim3(NB_THREADS), 0, stream[s], d_inBuffer + s*inOffset ,d_outBuffer + s*outOffset);
	}
	checkCUDAError(" ker_keccak");

	
	//copy back device to host	
	hipMemcpy(h_outBuffer,d_outBuffer, OUTPUT_BLOCK_SIZE_B * NB_THREADS*NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
	//checkCUDAError(" Memcpy dth");


	for(s=0;s<NB_STREAMS;s++)
	{hipStreamDestroy(stream[s]);}
	checkCUDAError(" StreamDestroy");

}


//***********************************************************
//Split computation over several Cuda STREAMS, and overlap with CPU
//***********************************************************
extern "C" 
__host__  void KeccakTreeGPU_Stream_OverlapCPU(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
								   tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer, 
								   tKeccakLane *Kstate)
{
	unsigned int inOffset;
	unsigned int outOffset;
	int s;

	hipStream_t stream[NB_STREAMS];
	
	for(s=0;s<NB_STREAMS;s++)
	{hipStreamCreate(&stream[s]);}
	checkCUDAError(" StreamCreate");
	
	
	//Offset (for 32b Word ) 
	inOffset= INPUT_BLOCK_SIZE_B/4 * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS / NB_STREAMS ;
	outOffset = OUTPUT_BLOCK_SIZE_B/4 * NB_THREADS*NB_THREADS_BLOCKS / NB_STREAMS ;

	

	//exec kernels 			
	for(s=0;s<NB_STREAMS;s++)
	{
		//copy host to device, PARTIAL 
		hipMemcpyAsync(d_inBuffer + s* inOffset , h_inBuffer + s*inOffset , inOffset * 4 , hipMemcpyHostToDevice, stream[s]);		
		//checkCUDAError(" MemcpyAsync htd");

		hipLaunchKernelGGL(ker_Keccak, dim3(NB_THREADS_BLOCKS/NB_STREAMS ), dim3(NB_THREADS), 0, stream[s], d_inBuffer + s*inOffset ,d_outBuffer + s*outOffset);
	}
	//checkCUDAError(" ker_keccak");

	Keccak_top_GPU(Kstate,h_outBuffer,NB_THREADS*NB_THREADS_BLOCKS);

	//hipDeviceSynchronize(); // should be useless when followed by hipMemcpy	
	
	//copy back device to host	
	hipMemcpy(h_outBuffer,d_outBuffer, OUTPUT_BLOCK_SIZE_B * NB_THREADS*NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
	//checkCUDAError(" Memcpy dth");


	for(s=0;s<NB_STREAMS;s++)
	{hipStreamDestroy(stream[s]);}
	checkCUDAError(" StreamDestroy");

}


//******************************************************
//2 stages Stage
//******************************************************

extern "C" 
__host__  void KeccakTreeGPU_2stg(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
							 tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer)
{
  //copy host to device 
  hipMemcpy(d_inBuffer,h_inBuffer, INPUT_BLOCK_SIZE_B * NB_THREADS * NB_INPUT_BLOCK*NB_THREADS_BLOCKS, hipMemcpyHostToDevice);		
  //checkCUDAError(" Memcpy htd");

  //exec kernels 			
  
  hipLaunchKernelGGL(ker_Keccak_2stg, dim3(NB_THREADS_BLOCKS), dim3(NB_THREADS), 0, 0, d_inBuffer,d_outBuffer);
  //checkCUDAError(" ker_keccak_2stg");


  
  //hipDeviceSynchronize(); // should be useless when followed by hipMemcpy
		
  //copy back device to host	
  hipMemcpy(h_outBuffer,d_outBuffer, 2*OUTPUT_BLOCK_SIZE_B * NB_SCND_STAGE_THREADS * NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
  //checkCUDAError(" Memcpy dth");
}


//***********************************************************
//2 stages +  Split computation over several Cuda STREAMS, and overlap with CPU
//***********************************************************
extern "C" 
__host__  void KeccakTreeGPU_2stg_Stream_OverlapCPU(tKeccakLane * h_inBuffer, tKeccakLane * d_inBuffer,
								   tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer, 
								   tKeccakLane *Kstate)
{
	unsigned int inOffset;
	unsigned int outOffset;
	int s;

	hipStream_t stream[NB_STREAMS];
	
	for(s=0;s<NB_STREAMS;s++)
	{hipStreamCreate(&stream[s]);}
	checkCUDAError(" StreamCreate");
	
	
	//Offset (for 32b Word ) 
	inOffset= INPUT_BLOCK_SIZE_B/4 * NB_THREADS * NB_INPUT_BLOCK*  NB_THREADS_BLOCKS / NB_STREAMS ;
	outOffset = OUTPUT_BLOCK_SIZE_B/4 * 2* NB_SCND_STAGE_THREADS *  NB_THREADS_BLOCKS / NB_STREAMS ;

	
	//exec kernels 			
	for(s=0;s<NB_STREAMS;s++)
	{
		//copy host to device, PARTIAL 
		hipMemcpyAsync(d_inBuffer + s* inOffset , h_inBuffer + s*inOffset , inOffset * 4 , hipMemcpyHostToDevice, stream[s]);		
		//checkCUDAError(" MemcpyAsync htd");

		hipLaunchKernelGGL(ker_Keccak_2stg, dim3(NB_THREADS_BLOCKS/NB_STREAMS ), dim3(NB_THREADS), 0, stream[s], d_inBuffer + s*inOffset ,d_outBuffer + s*outOffset);
	}
	//checkCUDAError(" ker_keccak_2stg streams");

	Keccak_top_GPU(Kstate,h_outBuffer,2* NB_SCND_STAGE_THREADS * NB_THREADS_BLOCKS);

	//hipDeviceSynchronize(); // should be useless when followed by hipMemcpy	
	
	//copy back device to host	
	hipMemcpy(h_outBuffer,d_outBuffer, 2*OUTPUT_BLOCK_SIZE_B * NB_SCND_STAGE_THREADS * NB_THREADS_BLOCKS ,hipMemcpyDeviceToHost);
	//checkCUDAError(" Memcpy dth");

	for(s=0;s<NB_STREAMS;s++)
	{hipStreamDestroy(stream[s]);}
	checkCUDAError(" StreamDestroy");

}


//***********************************************************
//Keccak in StreamCipher Mode  (Using Cuda Streams)
//***********************************************************
extern "C" 
__host__  void KeccakSCipherGPU_Stream(tKeccakLane * h_inKeyNonce, tKeccakLane * d_inKeyNonce,
								   tKeccakLane * h_outBuffer, tKeccakLane * d_outBuffer)
{
	unsigned int inOffset;
	unsigned int outOffset;
	int s;

	hipStream_t stream[NB_STREAMS];
	
	for(s=0;s<NB_STREAMS;s++)
	{hipStreamCreate(&stream[s]);}
	checkCUDAError(" StreamCreate");
		
	//Offset (for 32b Word ) 
	outOffset = SC_NB_OUTPUT_BLOCK * OUTPUT_BLOCK_SIZE_B/4 * NB_THREADS*NB_THREADS_BLOCKS / NB_STREAMS ;

	//copy Key and Nonce from Host
	hipMemcpy(d_inKeyNonce, h_inKeyNonce, 2* (32) ,hipMemcpyHostToDevice);
	checkCUDAError(" hipMemcpy HtD");

	//exec kernels 			
	for(s=0;s<NB_STREAMS;s++)
	{
		hipLaunchKernelGGL(ker_Keccak_SCipher, dim3(NB_THREADS_BLOCKS/NB_STREAMS ), dim3(NB_THREADS), 0, stream[s], d_inKeyNonce ,d_outBuffer + s*outOffset);
		checkCUDAError(" ker_keccak_SCipher");

		//copy Device to Host, PARTIAL 
		hipMemcpyAsync(h_outBuffer + s* outOffset , d_outBuffer + s*outOffset ,
			outOffset * 4 /*size in Byte*/ ,
			hipMemcpyDeviceToHost, stream[s]);

		checkCUDAError(" MemcpyAsync DtH");
	}


	for(s=0;s<NB_STREAMS;s++)
	{hipStreamDestroy(stream[s]);}
	checkCUDAError(" StreamDestroy");

}



